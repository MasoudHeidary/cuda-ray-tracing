#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <cstdlib> // run command on terminal
#include <string>
#include <chrono>  // For high-resolution clock
#include <iomanip> // For setting precision
#include <thread>

#include "setting.h"
#include "command_line_tool.h"
#include "loader.h"
// #include "gobject.h"
#include "compute.h"

#include <opencv2/opencv.hpp>
#include <hip/hip_runtime.h>

using namespace GeoShape;
using namespace std::chrono; // For convenient timing functions

__global__ void render_image(
    Camera camera,
    glm::vec3 *image,
    const int image_width,
    const int image_height,
    Sphere *spheres,
    int num_spheres,
    Plane *planes,
    int num_plains,
    Triangle *triangles,
    int num_triangles,
    Light *lights,
    int num_lights,
    const glm::vec3 background)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < image_width && j < image_height)
    {
        float u = float(image_width - 1 - i) / float(image_width - 1);
        float v = float(image_height - 1 - j) / float(image_height - 1); // Flip the v coordinate

        Ray ray = camera.get_ray(u, v);

        glm::vec3 pixel_color = background;
        Hit closest_hit_record;
        closest_hit_record.hit = false;

        Hit hit_record;
        for (int i = 0; i < num_spheres; i++)
        {
            // if (intersect_sphere(ray, sphere.center, sphere.radius, hit_record)) {
            if (spheres[i].intersect(ray, hit_record))
            {
                if (hit_record.t < closest_hit_record.t)
                {
                    closest_hit_record = hit_record;
                }
            }
        }

        for (int i = 0; i < num_plains; i++)
        {
            if (planes[i].intersect(ray, hit_record))
            {
                if (hit_record.t < closest_hit_record.t)
                {
                    closest_hit_record = hit_record;
                }
            }
        }

        for (int i = 0; i < num_triangles; i++)
        {
            if (triangles[i].intersect(ray, hit_record))
            {
                if (hit_record.t < closest_hit_record.t)
                {
                    closest_hit_record = hit_record;
                }
            }
        }

        // Compute lighting and color based on the closest hit
        if (closest_hit_record.hit)
        {
            glm::vec3 point = ray.at(closest_hit_record.t);
            glm::vec3 lighting = compute_lighting(point, closest_hit_record.normal, lights, num_lights);
            pixel_color = glm::clamp(closest_hit_record.color * lighting, 0.0f, 1.0f);
        }

        image[j * image_width + i] = pixel_color;

    }
}

int _main(int argc, char *argv[])
{
    cArg::CommandLineArgs args;
    cArg::ErrorCode error_code = cArg::parse_arguments(&args, argc, argv);

    if (error_code)
    {
        if (error_code == cArg::ErrorCode::HELP_REQUEST)
        {
            std::cout << cArg::__help_str__() << std::endl;
            return 0;
        }
        else
        {
            std::cerr << "CommandLineArgs Error, " << cArg::get_error_description(error_code) << std::endl;
            return error_code;
        }
    }
    std::cout << cArg::__str__(args) << std::endl;

    const std::string scene_file_name = args.scene_file;
    const std::string obj_file_name = args.obj_file;
    const std::string output_file_name = args.out_file;
    const int image_width = args.width;
    const int image_height = args.height;
    const bool shadow_enable = args.shadow;
    const bool cuda_enable = args.cuda;
    const int num_threads = args.num_threads;

    cv::Mat image(image_height, image_width, CV_8UC3);
    glm::vec3 background(0.0f, 0.0f, 0.0f);
    Camera camera(glm::vec3(0, 0, 0), glm::vec3(0, 0, 0), glm::vec3(0, 0, 0), 0.0f);

    std::vector<Sphere> spheres;
    std::vector<Plane> planes;
    std::vector<Triangle> triangles;
    std::vector<Light> lights;

    if (!load_scene(scene_file_name, spheres, planes, camera, lights, background))
    {
        std::cerr << "Error loading scene!\t" << scene_file_name << std::endl;
        return -1;
    }

    if (obj_file_name != "")
    {
        if (!load_obj(obj_file_name, triangles, glm::vec3(1.0, 1.0, 1.0)))
        {
            std::cerr << "Error loading OBJ file!\t" << obj_file_name << std::endl;
            return -1;
        }
    }

    std::cout << "lights: " << lights.size() << std::endl;
    std::cout << "spheres: " << spheres.size() << std::endl;
    std::cout << "planes: " << planes.size() << std::endl;
    std::cout << "triangle: " << triangles.size() << std::endl;
    std::cout << std::endl;

    auto start_time = high_resolution_clock::now();
    auto end_time = high_resolution_clock::now();
    double total_render_time = duration<double>(end_time - start_time).count();

    // ==================== CUDA ====================
    if (cuda_enable)
    {
        start_time = high_resolution_clock::now();

        glm::vec3 *d_image;
        hipMalloc(&d_image, image_width * image_height * sizeof(glm::vec3));

        Light *d_lights;
        Sphere *d_spheres;
        Plane *d_planes;
        Triangle *d_triangles;
        hipMalloc(&d_lights, lights.size() * sizeof(Light));
        hipMalloc(&d_spheres, spheres.size() * sizeof(Sphere));
        hipMalloc(&d_planes, planes.size() * sizeof(Plane));
        hipMalloc(&d_triangles, triangles.size() * sizeof(Triangle));

        hipMemcpy(d_lights, lights.data(), lights.size() * sizeof(Light), hipMemcpyHostToDevice);
        hipMemcpy(d_spheres, spheres.data(), spheres.size() * sizeof(Sphere), hipMemcpyHostToDevice);
        hipMemcpy(d_planes, planes.data(), planes.size() * sizeof(Plane), hipMemcpyHostToDevice);
        hipMemcpy(d_triangles, triangles.data(), triangles.size() * sizeof(Triangle), hipMemcpyHostToDevice);

        dim3 threadsPerBlock(16, 16);
        dim3 numBlocks((image_width + 15) / 16, (image_height + 15) / 16);

        render_image<<<numBlocks, threadsPerBlock>>>(
            camera,
            d_image,
            image_width,
            image_height,
            d_spheres,
            spheres.size(),
            d_planes,
            planes.size(),
            d_triangles,
            triangles.size(),
            d_lights,
            lights.size(),
            background);

        hipDeviceSynchronize();

        glm::vec3 *h_image = new glm::vec3[image_width * image_height];
        hipMemcpy(h_image, d_image, image_width * image_height * sizeof(glm::vec3), hipMemcpyDeviceToHost);

        hipFree(d_image);
        hipFree(d_spheres);
        hipFree(d_planes);
        hipFree(d_triangles);
        hipFree(d_lights);

        end_time = high_resolution_clock::now();
        total_render_time = duration<double>(end_time - start_time).count();


        for (int j = 0; j < image_height; ++j)
        {
            for (int i = 0; i < image_width; ++i)
            {
                glm::vec3 color = h_image[j * image_width + i];
                image.at<cv::Vec3b>(j, i)[0] = static_cast<unsigned char>(255.0f * color.b);
                image.at<cv::Vec3b>(j, i)[1] = static_cast<unsigned char>(255.0f * color.g);
                image.at<cv::Vec3b>(j, i)[2] = static_cast<unsigned char>(255.0f * color.r);
            }
        }

        delete[] h_image;
    }

    // ==================== CPU ====================
    else
    {
        start_time = high_resolution_clock::now();

        // chunk render
        auto render_chunk = [&](int start_row, int end_row, int thread_id)
        {
            // Render loop
            for (int j = start_row; j < end_row; ++j)
            {
                for (int i = 0; i < image_width; ++i)
                {

                    //flip coordinate
                    float u = float(image_width - 1 - i) / float(image_width - 1);
                    float v = float(image_height - 1 - j) / (image_height - 1);

                    Ray ray = camera.get_ray(u, v);
                    glm::vec3 pixel_color = background;
                    Hit closest_hit_record;

                    // Iterate through each sphere to find the closest intersection
                    for (const Sphere &sphere : spheres)
                    {
                        Hit hit_record;
                        if (sphere.intersect(ray, hit_record))
                        {
                            if (hit_record.t < closest_hit_record.t)
                            {
                                closest_hit_record = hit_record;
                            }
                        }
                    }

                    // Check plane intersections
                    for (const Plane &plane : planes)
                    {
                        Hit hit_record;
                        if (plane.intersect(ray, hit_record))
                        {
                            if (hit_record.t < closest_hit_record.t)
                            {
                                closest_hit_record = hit_record;
                            }
                        }
                    }

                    // Check triangle intersections
                    for (const Triangle &triangle : triangles)
                    {
                        Hit hit_record;
                        if (triangle.intersect(ray, hit_record))
                        {
                            if (hit_record.t < closest_hit_record.t)
                            {
                                closest_hit_record = hit_record;
                            }
                        }
                    }

                    // Compute lighting and color based on the closest hit
                    if (closest_hit_record.hit)
                    {
                        glm::vec3 point = ray.at(closest_hit_record.t);

                        glm::vec3 lighting;
                        if (shadow_enable)
                        {
                            // lighting = compute_lighting_and_shadow(point, closest_hit_record.normal, lights, spheres);
                        }
                        else
                        {
                            lighting = compute_lighting(point, closest_hit_record.normal, lights);
                        }

                        pixel_color = glm::clamp(closest_hit_record.color * lighting, 0.0f, 1.0f);
                    }
                    else
                    {
                        pixel_color = background; // If no intersection, use the background color
                    }

                    image.at<cv::Vec3b>(j, i)[0] = static_cast<unsigned char>(255.0 * pixel_color.b); 
                    image.at<cv::Vec3b>(j, i)[1] = static_cast<unsigned char>(255.0 * pixel_color.g); 
                    image.at<cv::Vec3b>(j, i)[2] = static_cast<unsigned char>(255.0 * pixel_color.r); 
                }
            }

            // end of chunk render
        };

        // Launch threads
        std::vector<std::thread> threads;
        int rows_per_thread = image_height / num_threads;
        for (unsigned int t = 0; t < num_threads; ++t)
        {
            int start_row = t * rows_per_thread;
            int end_row = (t == num_threads - 1) ? image_height : start_row + rows_per_thread;
            threads.push_back(std::thread(render_chunk, start_row, end_row, t));
        }

        // Join threads
        for (std::thread &thread : threads)
        {
            if (thread.joinable())
            {
                thread.join();
            }
        }

        end_time = high_resolution_clock::now();
        total_render_time = duration<double>(end_time - start_time).count();
    }



    std::cout << std::fixed << std::setprecision(6);
    std::cout << "Total render time: " << total_render_time << " seconds" << std::endl;

    if (cv::imwrite(output_file_name, image))
    {
        std::cout << "Image saved successfully to " << output_file_name << std::endl;
    }
    else
    {
        std::cerr << "Error: Could not save the image!" << std::endl;
    }

    return 0;
}